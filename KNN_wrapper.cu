#include <stdio.h>
#include <stdlib.h>
#include "KNN.h"
#include <time.h>
//#include <jni.h>
#include "hip/hip_runtime.h"
#include "knn_cuda_with_indexes.cu"

JNIEXPORT jint JNICALL Java_KNN_KNN_1search
  (JNIEnv *env, jobject obj, jint ref_nb, jint query_nb, jint dim, jint k) {
	// Variables and parameters
    float* ref;                 // Pointer to reference point array
    float* query;               // Pointer to query point array
    float* dist;                // Pointer to distance array
	int*   ind;                 // Pointer to index array
	int    iterations = 100;
	int    i;
	
	// Memory allocation
	ref    = (float *) malloc(ref_nb   * dim * sizeof(float));
	query  = (float *) malloc(query_nb * dim * sizeof(float));
	dist   = (float *) malloc(query_nb * k * sizeof(float));
	ind    = (int *)   malloc(query_nb * k * sizeof(float));
	
	// Init 
	srand(time(NULL));
	for (i=0 ; i<ref_nb   * dim ; i++) ref[i]    = (float)rand() / (float)RAND_MAX;
	for (i=0 ; i<query_nb * dim ; i++) query[i]  = (float)rand() / (float)RAND_MAX;
	
	// Variables for duration evaluation
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsed_time;
	
	// Display informations
	printf("Number of reference points      : %6d\n", ref_nb  );
	printf("Number of query points          : %6d\n", query_nb);
	printf("Dimension of points             : %4d\n", dim     );
	printf("Number of neighbors to consider : %4d\n", k       );
	printf("Processing kNN search           :"                );
	
	// Call kNN search CUDA
	hipEventRecord(start, 0);
	for (i=0; i<iterations; i++)
		knn(ref, ref_nb, query, query_nb, dim, k, dist, ind);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf(" done in %f s for %d iterations (%f s by iteration)\n", elapsed_time/1000, iterations, elapsed_time/(iterations*1000));
	
	// Destroy cuda event object and free memory
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(ind);
	free(dist);
	free(query);
	free(ref);

return 0;
  }
  

int main() {
return 0;
}
